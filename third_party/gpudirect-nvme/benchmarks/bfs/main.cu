#include "hip/hip_runtime.h"
/* References:
 *
 *      Baseline
 *          Harish, Pawan, and P. J. Narayanan.
 *          "Accelerating large graph algorithms on the GPU using CUDA."
 *          International conference on high-performance computing.
 *          Springer, Berlin, Heidelberg, 2007.
 *
 *      Coalesce
 *          Hong, Sungpack, et al.
 *          "Accelerating CUDA graph algorithms at maximum warp."
 *          Acm Sigplan Notices 46.8 (2011): 267-276.
 *
 */

#include <hip/hip_runtime.h>
#include <fstream>
#include <stdint.h>
#include <stdio.h>
#include <iostream>
#include <string.h>
#include <getopt.h>
//#include "hip/hip_runtime_api.h"
#include <algorithm>
#include <vector>
#include <numeric>
#include <iterator>
#include <math.h>
#include <chrono>
#include <ctime>
#include <ratio>
#include <cstdio>
#include <cstdint>
#include <cstring>
#include <fcntl.h>
#include <unistd.h>
#include <sys/mman.h>
#include <stdexcept>

#include <nvm_ctrl.h>
#include <nvm_types.h>
#include <nvm_queue.h>
#include <nvm_util.h>
#include <nvm_admin.h>
#include <nvm_error.h>
#include <nvm_cmd.h>
#include <buffer.h>
#include "settings.h"
#include <ctrl.h>
#include <event.h>
#include <queue.h>
#include <nvm_parallel_queue.h>
#include <nvm_io.h>
#include <page_cache.h>
#include <util.h>
#include <chrono>
#include <iostream>

using error = std::runtime_error;
using std::string;
//const char* const ctrls_paths[] = {"/dev/libnvm0","/dev/libnvm1",   "/dev/libnvm2", "/dev/libnvm3", "/dev/libnvm4", "/dev/libnvm5", "/dev/libnvm6", "/dev/libnvm7", "/dev/libnvm8", "/dev/libnvm9"};
const char* const ctrls_paths[] = {"/dev/libnvm0", "/dev/libnvm1", "/dev/libnvm4", "/dev/libnvm9", "/dev/libnvm2", "/dev/libnvm3", "/dev/libnvm5", "/dev/libnvm6", "/dev/libnvm7", "/dev/libnvm8"};
//const char* const ctrls_paths[] = {"/dev/libnvm0"};

#define MYINFINITY 0xFFFFFFFF

#define WARP_SHIFT 5
#define WARP_SIZE 32

#define CHUNK_SHIFT 3
#define CHUNK_SIZE (1 << CHUNK_SHIFT)

#define BLOCK_NUM 1024ULL

typedef uint64_t EdgeT;

typedef enum {
    BASELINE = 0,
    COALESCE = 1,
    COALESCE_CHUNK = 2,
    BASELINE_PC = 3,
    COALESCE_PC = 4, 
    COALESCE_CHUNK_PC = 5,
    FRONTIER_BASELINE = 6,
    FRONTIER_COALESCE = 7,
    FRONTIER_BASELINE_PC = 8,
    FRONTIER_COALESCE_PC = 9,
    BASELINE_PTR_PC = 10,
    COALESCE_PTR_PC = 11,
    COALESCE_CHUNK_PTR_PC = 12,
    FRONTIER_BASELINE_PTR_PC = 13,
    FRONTIER_COALESCE_PTR_PC = 14,
    COALESCE_HASH = 15,
    COALESCE_HASH_PTR_PC = 16,
} impl_type;

typedef enum {
    GPUMEM = 0,
    UVM_READONLY = 1,
    UVM_DIRECT = 2,
    UVM_READONLY_NVLINK = 3,
    UVM_DIRECT_NVLINK = 4,
    DRAGON_MAP = 5,
    BAFS_DIRECT = 6,
} mem_type;

__global__ __launch_bounds__(128,16)
void kernel_frontier_baseline(unsigned int *label, const unsigned int level, const uint64_t vertex_count,
                                const uint64_t *vertexList, const EdgeT *edgeList, const uint64_t curr_frontier_size, unsigned long long int *changed,
                                const uint32_t *curr_frontier, uint32_t *next_frontier) {
    //const uint64_t tid = blockDim.x * BLOCK_NUM * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
    const uint64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    //const uint64_t warpIdx = tid >> WARP_SHIFT;
    const uint32_t laneIdx = tid &  ((1 << WARP_SHIFT) - 1);

    if (tid < curr_frontier_size) {
        const uint32_t nid = curr_frontier[tid];
        const uint64_t start = vertexList[nid];
        const uint64_t end = vertexList[nid+1];

        for (uint64_t i = start; i < end; i++) {
            const EdgeT next = edgeList[i];

            if(label[next] == MYINFINITY) {
                //unsigned int prev = atomicExch(label+next, level+1);
                //if (prev == MYINFINITY) {
                    //performance code
                    // unsigned int pre_val = atomicCAS(&(label[next]),(unsigned int)MYINFINITY,(unsigned int)(level+1));
                    // if(pre_val == MYINFINITY){
                    //     atomicAdd(&globalvisitedcount_d[0], (unsigned long long int)(vertexList[next+1] - vertexList[next]));
                    // }
                    // *changed = true;

                    // uint32_t mask = __activemask();

                    // int leader = __ffs(mask) - 1;
                    // unsigned long long int pos;
                    // if (laneIdx == leader) {
                    //     pos = atomicAdd(changed, (unsigned long long int)__popc(mask));

                    // }
                    label[next] = level + 1;
                    uint64_t mypos = atomicAdd(changed, 1);
                    //pos = __shfl_sync(mask, pos, leader);

                    //unsigned long long int mypos = (pos) + __popc(mask & ((1 << laneIdx) - 1));

                    next_frontier[mypos] = next;

                //}

            }
        }
    }


}

__global__ __launch_bounds__(128,16)
void kernel_frontier_coalesce(unsigned int *label, const unsigned int level, const uint64_t vertex_count,
                                const uint64_t *vertexList, const EdgeT *edgeList, const uint64_t curr_frontier_size, unsigned long long int *changed,
                                const uint32_t *curr_frontier, uint32_t *next_frontier) {
    //const uint64_t tid = blockDim.x * BLOCK_NUM * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
    const uint64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    const uint64_t warpIdx = tid >> WARP_SHIFT;
    const uint64_t laneIdx = tid & ((1 << WARP_SHIFT) - 1);

    if (warpIdx < curr_frontier_size) {
        const uint32_t nid = curr_frontier[warpIdx];
        const uint64_t start = vertexList[nid];
        const uint64_t shift_start = start & 0xFFFFFFFFFFFFFFF0;
        const uint64_t end = vertexList[nid+1];


        for (uint64_t i = shift_start + laneIdx; i < end; i += WARP_SIZE) {
            if (i >= start) {
                const EdgeT next = edgeList[i];

                if(label[next] == MYINFINITY) {
                    uint32_t prev = atomicExch(label+next, level+1);
                    if (prev == MYINFINITY) {


                        //label[next] = level + 1;
                        uint32_t mask = __activemask();
                        uint32_t leader = __ffs(mask) - 1;
                        unsigned long long pos;
                        if (laneIdx == leader)
                            pos = atomicAdd(changed, __popc(mask));
                        pos = __shfl_sync(mask, pos, leader);
                        uint64_t mypos = pos + __popc(mask & ((1 << laneIdx) - 1));

                        //uint64_t mypos = atomicAdd(changed, 1);
                        next_frontier[mypos] = next;
                    }

                }
            }
        }
    }


}

__global__ __launch_bounds__(128,16)
void kernel_frontier_baseline_pc(unsigned int *label, const unsigned int level, const uint64_t vertex_count,
                                const uint64_t *vertexList, array_d_t<uint64_t>* da, const uint64_t curr_frontier_size, unsigned long long int *changed,
                                const uint32_t *curr_frontier, uint32_t *next_frontier) {
    //const uint64_t tid = blockDim.x * BLOCK_NUM * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
    const uint64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    //const uint64_t warpIdx = tid >> WARP_SHIFT;
    const uint32_t laneIdx = tid &  ((1 << WARP_SHIFT) - 1);

    if (tid < curr_frontier_size) {
        const uint32_t nid = curr_frontier[tid];
        const uint64_t start = vertexList[nid];
        const uint64_t end = vertexList[nid+1];

        for (uint64_t i = start; i < end; i++) {
            const EdgeT next = da->seq_read(i);

            if(label[next] == MYINFINITY) {
                unsigned int prev = atomicExch(label+next, level+1);
                if (prev == MYINFINITY) {
                    //performance code
                    // unsigned int pre_val = atomicCAS(&(label[next]),(unsigned int)MYINFINITY,(unsigned int)(level+1));
                    // if(pre_val == MYINFINITY){
                    //     atomicAdd(&globalvisitedcount_d[0], (unsigned long long int)(vertexList[next+1] - vertexList[next]));
                    // }
                    // *changed = true;

                    uint32_t mask = __activemask();

                    int leader = __ffs(mask) - 1;
                    unsigned long long int pos;
                    if (laneIdx == leader) {
                        pos = atomicAdd(changed, (unsigned long long int)__popc(mask));

                    }
                    label[next] = level + 1;
                    //uint64_t mypos = atomicAdd(changed, 1);
                    pos = __shfl_sync(mask, pos, leader);

                    unsigned long long int mypos = (pos) + __popc(mask & ((1 << laneIdx) - 1));

                    next_frontier[mypos] = next;

                }

            }
        }
    }


}

__global__ __launch_bounds__(128,16)
void kernel_frontier_coalesce_pc(unsigned int *label, const unsigned int level, const uint64_t vertex_count,
                                const uint64_t *vertexList, array_d_t<uint64_t>* da, const uint64_t curr_frontier_size, unsigned long long int *changed,
                                const uint32_t *curr_frontier, uint32_t *next_frontier) {
    //const uint64_t tid = blockDim.x * BLOCK_NUM * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
    const uint64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    const uint64_t warpIdx = tid >> WARP_SHIFT;
    const uint64_t laneIdx = tid & ((1 << WARP_SHIFT) - 1);

    if (warpIdx < curr_frontier_size) {
        const uint32_t nid = curr_frontier[warpIdx];
        const uint64_t start = vertexList[nid];
        const uint64_t shift_start = start & 0xFFFFFFFFFFFFFFF0;
        const uint64_t end = vertexList[nid+1];


        for (uint64_t i = shift_start + laneIdx; i < end; i += WARP_SIZE) {
            if (i >= start) {
                const EdgeT next = da->seq_read(i);

                if(label[next] == MYINFINITY) {
                    uint32_t prev = atomicExch(label+next, level+1);
                    if (prev == MYINFINITY) {


                        //label[next] = level + 1;
                        uint32_t mask = __activemask();
                        uint32_t leader = __ffs(mask) - 1;
                        unsigned long long pos;
                        if (laneIdx == leader)
                            pos = atomicAdd(changed, __popc(mask));
                        pos = __shfl_sync(mask, pos, leader);
                        uint64_t mypos = pos + __popc(mask & ((1 << laneIdx) - 1));

                        //uint64_t mypos = atomicAdd(changed, 1);
                        next_frontier[mypos] = next;
                    }

                }
            }
        }
    }
}


__global__ __launch_bounds__(128,16)
void kernel_frontier_coalesce_ptr_pc(unsigned int *label, const unsigned int level, const uint64_t vertex_count,
                                const uint64_t *vertexList, array_d_t<uint64_t>* da, const uint64_t curr_frontier_size, unsigned long long int *changed,
                                const uint32_t *curr_frontier, uint32_t *next_frontier) {
    //const uint64_t tid = blockDim.x * BLOCK_NUM * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
    const uint64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    const uint64_t warpIdx = tid >> WARP_SHIFT;
    const uint64_t laneIdx = tid & ((1 << WARP_SHIFT) - 1);

    if (warpIdx < curr_frontier_size) {
        bam_ptr<uint64_t> ptr(da);
        const uint32_t nid = curr_frontier[warpIdx];
        const uint64_t start = vertexList[nid];
        const uint64_t shift_start = start & 0xFFFFFFFFFFFFFFF0;
        const uint64_t end = vertexList[nid+1];


        for (uint64_t i = shift_start + laneIdx; i < end; i += WARP_SIZE) {
            if (i >= start) {
                //const EdgeT next = da->seq_read(i);
                const EdgeT next = ptr[i];

                if(label[next] == MYINFINITY) {
                    uint32_t prev = atomicExch(label+next, level+1);
                    if (prev == MYINFINITY) {
                        //label[next] = level + 1;
                        uint32_t mask = __activemask();
                        uint32_t leader = __ffs(mask) - 1;
                        unsigned long long pos;
                        if (laneIdx == leader)
                            pos = atomicAdd(changed, __popc(mask));
                        pos = __shfl_sync(mask, pos, leader);
                        uint64_t mypos = pos + __popc(mask & ((1 << laneIdx) - 1));

                        //uint64_t mypos = atomicAdd(changed, 1);
                        next_frontier[mypos] = next;
                    }

                }
            }
        }
    }
}



__global__ void kernel_baseline(uint32_t *label, const uint32_t level, const uint64_t vertex_count, 
                        const uint64_t *vertexList, const EdgeT *edgeList, uint64_t *changed, unsigned long long int *globalvisitedcount_d, unsigned long long int *vertexVisitCount_d
    ) {
    const uint64_t tid = blockDim.x * BLOCK_NUM * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;

    // if(tid==0)
    //         printf("Warning: The code is not optimal because of additional counters added for profiling\n");

    if(tid < vertex_count && label[tid] == level) {
        const uint64_t start = vertexList[tid];
        const uint64_t end = vertexList[tid+1];

        for(uint64_t i = start; i < end; i++) {
            const EdgeT next = edgeList[i];
            //performance code
            // atomicAdd(&vertexVisitCount_d[next], 1);

            if(label[next] == MYINFINITY) {
                //performance code
                // unsigned int pre_val = atomicCAS(&(label[next]),(unsigned int)MYINFINITY,(unsigned int)(level+1));
                // if(pre_val == MYINFINITY){
                //     atomicAdd(&globalvisitedcount_d[0], (unsigned long long int)(vertexList[next+1] - vertexList[next]));
                // }
                // *changed = true;

                label[next] = level + 1;
                *changed = true;
            }
        }
    }
}



__global__ __launch_bounds__(128,16)
void kernel_baseline_pc(array_d_t<uint64_t>* da, uint32_t *label, const uint32_t level, const uint64_t vertex_count,
                        const uint64_t *vertexList, const EdgeT *edgeList, uint64_t *changed, unsigned long long int *globalvisitedcount_d, unsigned long long int *vertexVisitCount_d
    ) {
    const uint64_t tid = blockDim.x * BLOCK_NUM * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;

//    array_d_t<uint64_t> d_array = *da;
    // if(tid==0)
    //         printf("Warning: The code is not optimal because of additional counters added for profiling\n");

    if(tid < vertex_count && label[tid] == level) {
        const uint64_t start = vertexList[tid];
        const uint64_t end = vertexList[tid+1];

        for(uint64_t i = start; i < end; i++) {
            //EdgeT next = da->seq_read(i);
            EdgeT next = da->seq_read(i);
//                printf("tid: %llu, idx: %llu next: %llu\n", (unsigned long long) tid, (unsigned long long) i, (unsigned long long) next);
            //performance code
            // atomicAdd(&vertexVisitCount_d[next], 1);

            if(label[next] == MYINFINITY) {
                //performance code
                // unsigned int pre_val = atomicCAS(&(label[next]),(unsigned int)MYINFINITY,(unsigned int)(level+1));
                // if(pre_val == MYINFINITY){
                //     atomicAdd(&globalvisitedcount_d[0], (unsigned long long int)(vertexList[next+1] - vertexList[next]));
                // }
                // *changed = true;

                label[next] = level + 1;
                *changed = true;
            }
        }
    }
}





__global__ void kernel_coalesce(uint32_t *label, const uint32_t level, const uint64_t vertex_count, const uint64_t *vertexList, const EdgeT *edgeList, uint64_t *changed) {
    const uint64_t tid = blockDim.x * BLOCK_NUM * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
    const uint64_t warpIdx = tid >> WARP_SHIFT;
    const uint64_t laneIdx = tid & ((1 << WARP_SHIFT) - 1);
    
    if(warpIdx < vertex_count && label[warpIdx] == level) {
        const uint64_t start = vertexList[warpIdx];
        const uint64_t shift_start = start & 0xFFFFFFFFFFFFFFF0;
        const uint64_t end = vertexList[warpIdx+1];

        for(uint64_t i = shift_start + laneIdx; i < end; i += WARP_SIZE) {
//        printf("Inside kernel %llu %llu %llu\n", (unsigned long long) i, (unsigned long long)start, (unsigned long long) (end-start));

            if (i >= start) {
                const EdgeT next = edgeList[i];
  //printf("tid: %llu, idx: %llu next: %llu\n", (unsigned long long) tid, (unsigned long long) i, (unsigned long long) next);

                if(label[next] == MYINFINITY) {

                //    if(level ==0)
                //            printf("tid:%llu, level:%llu, next: %llu start:%llu end:%llu\n", tid, (unsigned long long)level, (unsigned long long)next, (unsigned long long)start, (unsigned long long)end);
                    label[next] = level + 1;
                    *changed = true;
                }
            }
        }
    }
}

__global__ void kernel_coalesce_hash(uint32_t *label, const uint32_t level, const uint64_t vertex_count, const uint64_t *vertexList, const EdgeT *edgeList, uint64_t *changed, uint64_t stride) {
    const uint64_t oldtid = blockDim.x * BLOCK_NUM * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
    const uint64_t oldwarpIdx = oldtid >> WARP_SHIFT;
    const uint64_t laneIdx = oldtid & ((1 << WARP_SHIFT) - 1);
    uint64_t STRIDE = stride; 
    
    const uint64_t nep = (vertex_count+(STRIDE))/(STRIDE); 
    uint64_t warpIdx = (oldwarpIdx/nep) + ((oldwarpIdx % nep)*(STRIDE));
    
    if(warpIdx < vertex_count && label[warpIdx] == level) {
        const uint64_t start = vertexList[warpIdx];
        const uint64_t shift_start = start & 0xFFFFFFFFFFFFFFF0;
        const uint64_t end = vertexList[warpIdx+1];

        for(uint64_t i = shift_start + laneIdx; i < end; i += WARP_SIZE) {
//        printf("Inside kernel %llu %llu %llu\n", (unsigned long long) i, (unsigned long long)start, (unsigned long long) (end-start));

            if (i >= start) {
                const EdgeT next = edgeList[i];
  //printf("tid: %llu, idx: %llu next: %llu\n", (unsigned long long) tid, (unsigned long long) i, (unsigned long long) next);

                if(label[next] == MYINFINITY) {

                //    if(level ==0)
                //            printf("tid:%llu, level:%llu, next: %llu start:%llu end:%llu\n", tid, (unsigned long long)level, (unsigned long long)next, (unsigned long long)start, (unsigned long long)end);
                    label[next] = level + 1;
                    *changed = true;
                }
            }
        }
    }
}



__global__ __launch_bounds__(128,16)
void kernel_coalesce_pc(array_d_t<uint64_t>* da, uint32_t *label, const uint32_t level, const uint64_t vertex_count, const uint64_t *vertexList, const EdgeT *edgeList, uint64_t *changed) {
    const uint64_t tid = blockDim.x * BLOCK_NUM * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
    const uint64_t warpIdx = tid >> WARP_SHIFT;
    const uint64_t laneIdx = tid & ((1 << WARP_SHIFT) - 1);
    //array_d_t<uint64_t> d_array = *da;
    if(warpIdx < vertex_count && label[warpIdx] == level) {
        const uint64_t start = vertexList[warpIdx];
        const uint64_t shift_start = start & 0xFFFFFFFFFFFFFFF0;
        const uint64_t end = vertexList[warpIdx+1];

        for(uint64_t i = shift_start + laneIdx; i < end; i += WARP_SIZE) {
            if (i >= start) {
                //const EdgeT next = edgeList[i];
                //EdgeT next = da->seq_read(i);
                EdgeT next = da->seq_read(i);
//                printf("tid: %llu, idx: %llu next: %llu\n", (unsigned long long) tid, (unsigned long long) i, (unsigned long long) next);

                if(label[next] == MYINFINITY) {
                //    if(level ==0)
                //            printf("tid:%llu, level:%llu, next: %llu\n", tid, (unsigned long long)level, (unsigned long long)next);
                    label[next] = level + 1;
                    *changed = true;
                }
            }
        }
    }
}

__global__ __launch_bounds__(128,16)
void kernel_coalesce_ptr_pc(array_d_t<uint64_t>* da, uint32_t *label, const uint32_t level, const uint64_t vertex_count, const uint64_t *vertexList, const EdgeT *edgeList, uint64_t *changed) {
    const uint64_t tid = blockDim.x * BLOCK_NUM * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
    const uint64_t warpIdx = tid >> WARP_SHIFT;
    const uint64_t laneIdx = tid & ((1 << WARP_SHIFT) - 1);
    //array_d_t<uint64_t> d_array = *da;
    if(warpIdx < vertex_count && label[warpIdx] == level) {
        bam_ptr<uint64_t> ptr(da);
        const uint64_t start = vertexList[warpIdx];
        const uint64_t shift_start = start & 0xFFFFFFFFFFFFFFF0;
        const uint64_t end = vertexList[warpIdx+1];

        for(uint64_t i = shift_start + laneIdx; i < end; i += WARP_SIZE) {
            if (i >= start) {
                //const EdgeT next = edgeList[i];
                //EdgeT next = da->seq_read(i);
                EdgeT next = ptr[i];
//                printf("tid: %llu, idx: %llu next: %llu\n", (unsigned long long) tid, (unsigned long long) i, (unsigned long long) next);

                if(label[next] == MYINFINITY) {
                //    if(level ==0)
                //            printf("tid:%llu, level:%llu, next: %llu\n", tid, (unsigned long long)level, (unsigned long long)next);
                    label[next] = level + 1;
                    *changed = true;
                }
            }
        }
    }
}


__global__ __launch_bounds__(128,16)
void kernel_coalesce_hash_ptr_pc(array_d_t<uint64_t>* da, uint32_t *label, const uint32_t level, const uint64_t vertex_count, const uint64_t *vertexList, const EdgeT *edgeList, uint64_t *changed, uint64_t stride) {
    const uint64_t oldtid = blockDim.x * BLOCK_NUM * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
    const uint64_t oldwarpIdx = oldtid >> WARP_SHIFT;
    const uint64_t laneIdx = oldtid & ((1 << WARP_SHIFT) - 1);
    uint64_t STRIDE = stride; 
    const uint64_t nep = (vertex_count+(STRIDE))/(STRIDE); 
    uint64_t warpIdx = (oldwarpIdx/nep) + ((oldwarpIdx % nep)*(STRIDE));

    //array_d_t<uint64_t> d_array = *da;
    if(warpIdx < vertex_count && label[warpIdx] == level) {
        bam_ptr<uint64_t> ptr(da);
        const uint64_t start = vertexList[warpIdx];
        const uint64_t shift_start = start & 0xFFFFFFFFFFFFFFF0;
        const uint64_t end = vertexList[warpIdx+1];

        for(uint64_t i = shift_start + laneIdx; i < end; i += WARP_SIZE) {
            if (i >= start) {
                //const EdgeT next = edgeList[i];
                //EdgeT next = da->seq_read(i);
                EdgeT next = ptr[i];
//                printf("tid: %llu, idx: %llu next: %llu\n", (unsigned long long) tid, (unsigned long long) i, (unsigned long long) next);

                if(label[next] == MYINFINITY) {
                //    if(level ==0)
                //            printf("tid:%llu, level:%llu, next: %llu\n", tid, (unsigned long long)level, (unsigned long long)next);
                    label[next] = level + 1;
                    *changed = true;
                }
            }
        }
    }
}

__global__ void kernel_coalesce_chunk(uint32_t *label, const uint32_t level, const uint64_t vertex_count, const uint64_t *vertexList, const EdgeT *edgeList, uint64_t *changed) {
    const uint64_t tid = blockDim.x * BLOCK_NUM * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
    const uint64_t warpIdx = tid >> WARP_SHIFT;
    const uint64_t laneIdx = tid & ((1 << WARP_SHIFT) - 1);
    const uint64_t chunkIdx = warpIdx * CHUNK_SIZE;
    uint64_t chunk_size = CHUNK_SIZE;

    if((chunkIdx + CHUNK_SIZE) > vertex_count) {
        if ( vertex_count > chunkIdx )
            chunk_size = vertex_count - chunkIdx;
        else
            return;
    }

    for(uint32_t i = chunkIdx; i < chunk_size + chunkIdx; i++) {
        if(label[i] == level) {
            const uint64_t start = vertexList[i];
            const uint64_t shift_start = start & 0xFFFFFFFFFFFFFFF0;
            const uint64_t end = vertexList[i+1];

            for(uint64_t j = shift_start + laneIdx; j < end; j += WARP_SIZE) {
                if (j >= start) {
                    const EdgeT next = edgeList[j];
          
                    if(label[next] == MYINFINITY) {
                        label[next] = level + 1;
                        *changed = true;
                    }
                }
            }
        }
    }
}


__global__  __launch_bounds__(1024,2)
void kernel_coalesce_chunk_pc(array_d_t<uint64_t>* da, uint32_t *label, const uint32_t level, const uint64_t vertex_count, const uint64_t *vertexList, const EdgeT *edgeList, uint64_t *changed) {
    const uint64_t tid = blockDim.x * BLOCK_NUM * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
    const uint64_t warpIdx = tid >> WARP_SHIFT;
    const uint64_t laneIdx = tid & ((1 << WARP_SHIFT) - 1);
    const uint64_t chunkIdx = warpIdx * CHUNK_SIZE;
    uint64_t chunk_size = CHUNK_SIZE;
    //array_d_t<uint64_t> d_array = *da;
    if((chunkIdx + CHUNK_SIZE) > vertex_count) {
        if ( vertex_count > chunkIdx )
            chunk_size = vertex_count - chunkIdx;
        else
            return;
    }

    for(uint32_t i = chunkIdx; i < chunk_size + chunkIdx; i++) {
        if(label[i] == level) {
            const uint64_t start = vertexList[i];
            const uint64_t shift_start = start & 0xFFFFFFFFFFFFFFF0;
            const uint64_t end = vertexList[i+1];

            for(uint64_t j = shift_start + laneIdx; j < end; j += WARP_SIZE) {
                if (j >= start) {
                    // const EdgeT next = edgeList[j];
                    //EdgeT next = da->seq_read(j);
                    EdgeT next = da->seq_read(j);
                    // printf("tid: %llu, idx: %llu next: %llu\n", (unsigned long long) tid, (unsigned long long) i, (unsigned long long) next);

                    if(label[next] == MYINFINITY) {
                        label[next] = level + 1;
                        *changed = true;
                    }
                }
            }
        }
    }
}


__global__ void throttle_memory(uint32_t *pad) {
    pad[1] = pad[0];
}





int main(int argc, char *argv[]) {
    using namespace std::chrono; 

    Settings settings; 
    try
    {
        settings.parseArguments(argc, argv);
    }
    catch (const string& e)
    {
        fprintf(stderr, "%s\n", e.c_str());
        fprintf(stderr, "%s\n", Settings::usageString(argv[0]).c_str());
        return 1;
    }

    hipDeviceProp_t properties;
    if (hipGetDeviceProperties(&properties, settings.cudaDevice) != hipSuccess)
    {
        fprintf(stderr, "Failed to get CUDA device properties\n");
        return 1;
    }

    std::ifstream file;
    std::string vertex_file, edge_file;
    std::string filename;

    uint64_t changed_h, *changed_d;// no_src = false;
    int num_run = 0;// arg_num = 0;
    int total_run = 1;// arg_num = 0;
    impl_type type;
    mem_type mem;
    uint32_t *pad;
    uint32_t *label_d, level, zero, iter;
    uint64_t *vertexList_h, *vertexList_d;
    EdgeT *edgeList_h, *edgeList_d;
    uint64_t vertex_count, edge_count, vertex_size, edge_size;
    uint64_t typeT, src;
    uint64_t numblocks, numthreads;
    size_t freebyte, totalbyte;

    float milliseconds;
    double avg_milliseconds;

    uint64_t pc_page_size; 
    uint64_t pc_pages;
   
    try{
         //prepare from settings
         filename = std::string(settings.input); 

         if(settings.src == 0) {
                 total_run = settings.repeat; 
                 src = 0;
         }
         else {
                 total_run = 2; 
                 src = settings.src; 
         }

         type = (impl_type) settings.type; 
         mem = (mem_type) settings.memalloc; 

         pc_page_size = settings.pageSize; 
         pc_pages = ceil((float)settings.maxPageCacheSize/pc_page_size);

         numthreads = settings.numThreads;
         
         cuda_err_chk(hipSetDevice(settings.cudaDevice));
         
         hipEvent_t start, end;
         cuda_err_chk(hipEventCreate(&start));
         cuda_err_chk(hipEventCreate(&end));

         vertex_file = filename + ".col";
         edge_file = filename + ".dst";

         std::cout << filename << std::endl;
         fprintf(stderr, "File %s\n", filename.c_str());
         // Read files
         file.open(vertex_file.c_str(), std::ios::in | std::ios::binary);
         if (!file.is_open()) {
             fprintf(stderr, "Vertex file open failed\n");
             exit(1);
         };

         file.read((char*)(&vertex_count), 8);
         file.read((char*)(&typeT), 8);

         vertex_count--;

         printf("Vertex: %llu, ", vertex_count);
         vertex_size = (vertex_count+1) * sizeof(uint64_t);

         vertexList_h = (uint64_t*)malloc(vertex_size);

         file.read((char*)vertexList_h, vertex_size);
         file.close();

         file.open(edge_file.c_str(), std::ios::in | std::ios::binary);
         if (!file.is_open()) {
             fprintf(stderr, "Edge file open failed\n");
             exit(1);
         };

         file.read((char*)(&edge_count), 8);
         file.read((char*)(&typeT), 8);

         printf("Edge: %llu\n", edge_count);
         fflush(stdout);
         edge_size = edge_count * sizeof(EdgeT); //4096 padding for weights and edges. 
         edge_size = edge_size + (4096 - (edge_size & 0xFFFULL));

         edgeList_h = NULL;
         edgeList_d = NULL;

         // Allocate memory for GPU
         cuda_err_chk(hipMalloc((void**)&vertexList_d, vertex_size));
         cuda_err_chk(hipMalloc((void**)&label_d, vertex_count * sizeof(uint32_t)));
         cuda_err_chk(hipMalloc((void**)&changed_d, sizeof(uint64_t)));
     
         std::vector<unsigned long long int> vertexVisitCount_h;
         unsigned long long int* vertexVisitCount_d;
         unsigned long long int globalvisitedcount_h;
         unsigned long long int* globalvisitedcount_d;
     
         vertexVisitCount_h.resize(vertex_count);
         cuda_err_chk(hipMalloc((void**)&globalvisitedcount_d, sizeof(unsigned long long int)));
         cuda_err_chk(hipMemset(globalvisitedcount_d, 0, sizeof(unsigned long long int)));
         cuda_err_chk(hipMalloc((void**)&vertexVisitCount_d, vertex_count*sizeof(unsigned long long int)));
         cuda_err_chk(hipMemset(vertexVisitCount_d, 0, vertex_count*sizeof(unsigned long long int)));

         switch (mem) {
             case GPUMEM:
                 edgeList_h = (EdgeT*)malloc(edge_size);
                 file.read((char*)edgeList_h, edge_size);
                 cuda_err_chk(hipMalloc((void**)&edgeList_d, edge_size));
                 file.close();
                 break;
             case UVM_READONLY:
                 cuda_err_chk(hipMallocManaged((void**)&edgeList_d, edge_size));
                 file.read((char*)edgeList_d, edge_size);
                 cuda_err_chk(hipMemAdvise(edgeList_d, edge_size, hipMemAdviseSetReadMostly, settings.cudaDevice));
     
                 cuda_err_chk(hipMemGetInfo(&freebyte, &totalbyte));
                 if (totalbyte < 16*1024*1024*1024ULL)
                     printf("total memory sizeo of current GPU is %llu byte, no need to throttle\n", totalbyte);
                 else {
                     printf("total memory sizeo of current GPU is %llu byte, throttling %llu byte.\n", totalbyte, totalbyte - 16*1024*1024*1024ULL);
                     cuda_err_chk(hipMalloc((void**)&pad, totalbyte - 16*1024*1024*1024ULL));
                     throttle_memory<<<1,1>>>(pad);
                 }
                 file.close();
                 break;
             case UVM_DIRECT:
             {
             /*    cuda_err_chk(hipMallocManaged((void**)&edgeList_d, edge_size));
                 // printf("Address is %p   %p\n", edgeList_d, &edgeList_d[0]); 
                 high_resolution_clock::time_point ft1 = high_resolution_clock::now();
                 file.read((char*)edgeList_d, edge_size);
                 file.close();
                 high_resolution_clock::time_point ft2 = high_resolution_clock::now();
                 duration<double> time_span = duration_cast<duration<double>>(ft2 -ft1);
                 std::cout<< "edge file read time: "<< time_span.count() <<std::endl;
                 cuda_err_chk(hipMemAdvise(edgeList_d, edge_size, hipMemAdviseSetAccessedBy, settings.cudaDevice));
                 break;
             */

                 file.close();
                 for (uint64_t i = 0; i < vertex_count + 1; i++) {
                     vertexList_h[i] += 2;
                 }   
                 int fd = open(edge_file.c_str(), O_RDONLY | O_DIRECT);
                 FILE *file_temp = fdopen(fd, "rb");
                 if ((file_temp == NULL) || (fd == -1)) {
                     printf("edge file fd open failed\n");
                     exit(1);
                 }   
                 uint64_t edge_count_4k_aligned = ((edge_count + 2 + 4096 / sizeof(uint64_t)) / (4096 / sizeof(uint64_t))) * (4096 / sizeof(uint64_t));
                 uint64_t edge_size_4k_aligned = edge_count_4k_aligned * sizeof(uint64_t);
                 cuda_err_chk(hipMallocManaged((void**)&edgeList_d, edge_size_4k_aligned));
                 cuda_err_chk(hipMemAdvise(edgeList_d, edge_size_4k_aligned, hipMemAdviseSetAccessedBy, settings.cudaDevice));
                 high_resolution_clock::time_point ft1 = high_resolution_clock::now();
                       
                 if (fread(edgeList_d, sizeof(uint64_t), edge_count_4k_aligned, file_temp) != edge_count + 2) {
                     printf("edge file fread failed\n");
                     exit(1);
                 }   
                 fclose(file_temp);                                                                                                              
                 close(fd);
                 high_resolution_clock::time_point ft2 = high_resolution_clock::now();
                 duration<double> time_span = duration_cast<duration<double>>(ft2 -ft1);
                 std::cout<< "Edge file read time: "<< time_span.count() <<std::endl;
                       
                 file.open(edge_file.c_str(), std::ios::in | std::ios::binary);
                 if (!file.is_open()) {
                     printf("edge file open failed\n");
                     exit(1);
                 }   
                 break;
             }
             case BAFS_DIRECT: 
                 //cuda_err_chk(hipMemGetInfo(&freebyte, &totalbyte));
                 //if (totalbyte < 16*1024*1024*1024ULL)
                 //    printf("total memory sizeo of current GPU is %llu byte, no need to throttle\n", totalbyte);
                 //else {
                 //    printf("total memory sizeo of current GPU is %llu byte, throttling %llu byte.\n", totalbyte, totalbyte - 16*1024*1024*1024ULL);
                 //    cuda_err_chk(hipMalloc((void**)&pad, totalbyte - 16*1024*1024*1024ULL));
                 //    throttle_memory<<<1,1>>>(pad);
                 //}
                 break;
         }
     
     
         printf("Allocation finished\n");
         fflush(stdout);

         // Initialize values
         cuda_err_chk(hipMemcpy(vertexList_d, vertexList_h, vertex_size, hipMemcpyHostToDevice));

         if (mem == GPUMEM){
             cuda_err_chk(hipMemcpy(edgeList_d, edgeList_h, edge_size, hipMemcpyHostToDevice));
         }
    

         switch (type) {
             case BASELINE:
             case BASELINE_PC:
                 numblocks = ((vertex_count + numthreads) / numthreads);
                 break;
             case COALESCE:
             case COALESCE_HASH:
             case COALESCE_PC:
             case COALESCE_PTR_PC:
             case COALESCE_HASH_PTR_PC:
                 numblocks = ((vertex_count * WARP_SIZE + numthreads) / numthreads);
                 break;
             case COALESCE_CHUNK:
             case COALESCE_CHUNK_PC:
                 numblocks = ((vertex_count * (WARP_SIZE / CHUNK_SIZE) + numthreads) / numthreads);
                 break;
             case FRONTIER_BASELINE:
             case FRONTIER_COALESCE:
             case FRONTIER_BASELINE_PC:
             case FRONTIER_COALESCE_PC:
             case FRONTIER_COALESCE_PTR_PC:
                 break;
             default:
                 fprintf(stderr, "Invalid type\n");
                 exit(1);
                 break;
         }
    
         //TODO : FIX THIS. 
         dim3 blockDim(BLOCK_NUM, (numblocks+BLOCK_NUM)/BLOCK_NUM);

         avg_milliseconds = 0.0f;


         if((type==BASELINE_PC)||(type == COALESCE_PC) ||(type == COALESCE_CHUNK_PC)||(type==FRONTIER_BASELINE_PC)||(type == FRONTIER_COALESCE_PC) || (type== FRONTIER_COALESCE_PTR_PC)){
                printf("page size: %d, pc_entries: %llu\n", pc_page_size, pc_pages);
                fflush(stdout);
         }

         std::vector<Controller*> ctrls(settings.n_ctrls);
         if(mem == BAFS_DIRECT){
             cuda_err_chk(hipSetDevice(settings.cudaDevice));
             for (size_t i = 0 ; i < settings.n_ctrls; i++)
                 ctrls[i] = new Controller(ctrls_paths[i], settings.nvmNamespace, settings.cudaDevice, settings.queueDepth, settings.numQueues);
             printf("Controllers Created\n");
         }
         char gdevst[15];
         cuda_err_chk(hipDeviceGetPCIBusId(gdevst, 15, settings.cudaDevice));
         std::cout << "GPUID: "<< gdevst << std::endl;

         printf("Initialization done.\n");
         fflush(stdout);
         
         page_cache_t* h_pc; 
         range_t<uint64_t>* h_range;
         std::vector<range_t<uint64_t>*> vec_range(1);
         array_t<uint64_t>* h_array; 
         uint64_t n_pages = ceil(((float)edge_size)/pc_page_size);
         uint32_t* curr_frontier_d;
         uint32_t* next_frontier_d;
         
         if((type==BASELINE_PC)||(type == COALESCE_PC) ||(type == COALESCE_PTR_PC)||(type==COALESCE_HASH_PTR_PC) ||(type == COALESCE_CHUNK_PC)||(type==FRONTIER_BASELINE_PC)||(type == FRONTIER_COALESCE_PC)||(type==FRONTIER_COALESCE_PTR_PC)){
            h_pc =new page_cache_t(pc_page_size, pc_pages, settings.cudaDevice, ctrls[0][0], (uint64_t) 64, ctrls);
            h_range = new range_t<uint64_t>((uint64_t)0 ,(uint64_t)edge_count, (uint64_t) (ceil(settings.ofileoffset*1.0/pc_page_size)),(uint64_t)n_pages, (uint64_t)0, (uint64_t)pc_page_size, h_pc, settings.cudaDevice); //, (uint8_t*)edgeList_d);
            vec_range[0] = h_range; 
            h_array = new array_t<uint64_t>(edge_count, settings.ofileoffset, vec_range, settings.cudaDevice);
            
            printf("Page cache initialized\n");
            fflush(stdout);
         }
         if ((type==FRONTIER_BASELINE)||(type == FRONTIER_COALESCE) ||(type==FRONTIER_BASELINE_PC)||(type == FRONTIER_COALESCE_PC)||(type==FRONTIER_COALESCE_PTR_PC)){
             cuda_err_chk(hipMalloc((void**)&curr_frontier_d,  vertex_count * sizeof(uint32_t)));
             cuda_err_chk(hipMalloc((void**)&next_frontier_d,  vertex_count * sizeof(uint32_t)));
         }
         uint32_t* tmp_front;
         // Set root
         for (int i = 0; i < total_run; i++) {
             zero = 0;
             cuda_err_chk(hipMemset(label_d, 0xFF, vertex_count * sizeof(uint32_t)));
             cuda_err_chk(hipMemcpy(&label_d[src], &zero, sizeof(uint32_t), hipMemcpyHostToDevice));
             if ((type==FRONTIER_BASELINE)||(type == FRONTIER_COALESCE) ||(type==FRONTIER_BASELINE_PC)||(type == FRONTIER_COALESCE_PC)||(type==FRONTIER_COALESCE_PTR_PC)){
                 cuda_err_chk(hipMemcpy(curr_frontier_d, &src, sizeof(uint32_t), hipMemcpyHostToDevice));
             }

             level = 0;
             iter = 0;

             cuda_err_chk(hipEventRecord(start, 0));
   // printf("*****baseaddr: %p\n", h_pc->pdt.base_addr);
   //          fflush(stdout);

             // Run BFS
             changed_h = 1;

             printf("Hash Stride: %llu type:%llu\n", (settings.stride), type);
             
             do {
                 uint64_t active = changed_h;
                 changed_h = 0;
                 cuda_err_chk(hipMemcpy(changed_d, &changed_h, sizeof(uint64_t), hipMemcpyHostToDevice));
                 auto start = std::chrono::system_clock::now();
                 switch (type) {
                     case BASELINE:
                         kernel_baseline<<<blockDim, numthreads>>>(label_d, level, vertex_count, vertexList_d, edgeList_d, changed_d, globalvisitedcount_d, vertexVisitCount_d);
                         break;
                     case COALESCE:
                         kernel_coalesce<<<blockDim, numthreads>>>(label_d, level, vertex_count, vertexList_d, edgeList_d, changed_d);
                         break;
                     case COALESCE_HASH:
                         //TODO: fix the stride 
                         kernel_coalesce_hash<<<blockDim, numthreads>>>(label_d, level, vertex_count, vertexList_d, edgeList_d, changed_d, settings.stride);
                         break;
                     case COALESCE_CHUNK:
                         kernel_coalesce_chunk<<<blockDim, numthreads>>>(label_d, level, vertex_count, vertexList_d, edgeList_d, changed_d);
                         break;
                     case BASELINE_PC:
                         //printf("Calling Page cache enabled baseline kernel\n");
                         kernel_baseline_pc<<<blockDim, numthreads>>>(h_array->d_array_ptr, label_d, level, vertex_count, vertexList_d, edgeList_d, changed_d, globalvisitedcount_d, vertexVisitCount_d);
                         break;
                     case COALESCE_PC:
                         //printf("Calling Page cache enabled coalesce kernel\n");
                         kernel_coalesce_pc<<<blockDim, numthreads>>>(h_array->d_array_ptr, label_d, level, vertex_count, vertexList_d, edgeList_d, changed_d);
                         break;
                     case COALESCE_PTR_PC:
                         //printf("Calling Page cache enabled coalesce kernel\n");
                         kernel_coalesce_ptr_pc<<<blockDim, numthreads>>>(h_array->d_array_ptr, label_d, level, vertex_count, vertexList_d, edgeList_d, changed_d);
                         break;
                     case COALESCE_HASH_PTR_PC:
                         {
                             //TODO: fix the stride
                             //printf("Calling transposed kernel\n");
                             uint64_t stride = settings.stride; 
                             if(iter == 6){
                                 printf("changing stride\n");
                                 fflush(stdout);
                                 stride = 768; 
                             }
                             kernel_coalesce_hash_ptr_pc<<<blockDim, numthreads>>>(h_array->d_array_ptr, label_d, level, vertex_count, vertexList_d, edgeList_d, changed_d, stride);
                             break;
                         }
                    case COALESCE_CHUNK_PC:
                         //printf("Calling Page cache enabled coalesce chunk kernel\n");
                         kernel_coalesce_chunk_pc<<<blockDim, numthreads>>>(h_array->d_array_ptr, label_d, level, vertex_count, vertexList_d, edgeList_d, changed_d);
                         break;
                     case FRONTIER_BASELINE:
                          // kernel_frontier_baseline(uint32_t *label, const uint32_t level, const uint64_t vertex_count,
                          //       const uint64_t *vertexList, const EdgeT *edgeList, const uint64_t curr_frontier_size, unsigned long long *changed,
                          //                          const uint32_t *curr_frontier, uint32_t *next_frontier)
                         numblocks = ((active + numthreads) / numthreads);
                         assert(numblocks <= 0xFFFFFFFF);
                         kernel_frontier_baseline<<<numblocks, numthreads>>>((unsigned int*)label_d, (unsigned int) level, vertex_count, vertexList_d, edgeList_d, active,(unsigned long long int*)changed_d, curr_frontier_d, next_frontier_d);
                         tmp_front = curr_frontier_d;
                         curr_frontier_d = next_frontier_d;
                         next_frontier_d = tmp_front;
                         break;
                     case FRONTIER_COALESCE:
                         // kernel_frontier_baseline(uint32_t *label, const uint32_t level, const uint64_t vertex_count,
                         //       const uint64_t *vertexList, const EdgeT *edgeList, const uint64_t curr_frontier_size, unsigned long long *changed,
                         //                          const uint32_t *curr_frontier, uint32_t *next_frontier)
                         numblocks = ((active * WARP_SIZE + numthreads) / numthreads);
                         assert(numblocks <= 0xFFFFFFFF);
                         //printf("numblocks: %llu\n", numblocks);
                         kernel_frontier_coalesce<<<numblocks, numthreads>>>((unsigned int*)label_d, (unsigned int) level, vertex_count, vertexList_d, edgeList_d, active,(unsigned long long int*)changed_d, curr_frontier_d, next_frontier_d);
                         tmp_front = curr_frontier_d;
                         curr_frontier_d = next_frontier_d;
                         next_frontier_d = tmp_front;
                         break;
                     case FRONTIER_BASELINE_PC:
                          // kernel_frontier_baseline(uint32_t *label, const uint32_t level, const uint64_t vertex_count,
                          //       const uint64_t *vertexList, const EdgeT *edgeList, const uint64_t curr_frontier_size, unsigned long long *changed,
                          //                          const uint32_t *curr_frontier, uint32_t *next_frontier)
                         numblocks = ((active + numthreads) / numthreads);
                         assert(numblocks <= 0xFFFFFFFF);
                         kernel_frontier_baseline_pc<<<numblocks, numthreads>>>((unsigned int*)label_d, (unsigned int) level, vertex_count, vertexList_d, h_array->d_array_ptr, active,(unsigned long long int*)changed_d, curr_frontier_d, next_frontier_d);
                         tmp_front = curr_frontier_d;
                         curr_frontier_d = next_frontier_d;
                         next_frontier_d = tmp_front;
                         break;
                     case FRONTIER_COALESCE_PC:
                         // kernel_frontier_baseline(uint32_t *label, const uint32_t level, const uint64_t vertex_count,
                         //       const uint64_t *vertexList, const EdgeT *edgeList, const uint64_t curr_frontier_size, unsigned long long *changed,
                         //                          const uint32_t *curr_frontier, uint32_t *next_frontier)
                         numblocks = ((active * WARP_SIZE + numthreads) / numthreads);
                         assert(numblocks <= 0xFFFFFFFF);
                         //printf("numblocks: %llu\t", numblocks);
                         kernel_frontier_coalesce_pc<<<numblocks, numthreads>>>((unsigned int*)label_d, (unsigned int) level, vertex_count, vertexList_d, h_array->d_array_ptr, active,(unsigned long long int*)changed_d, curr_frontier_d, next_frontier_d);
                         tmp_front = curr_frontier_d;
                         curr_frontier_d = next_frontier_d;
                         next_frontier_d = tmp_front;
                         break;
                     case FRONTIER_COALESCE_PTR_PC:
                         numblocks = ((active * WARP_SIZE + numthreads) / numthreads);
                         assert(numblocks <= 0xFFFFFFFF);
                         //printf("numblocks: %llu\t", numblocks);
                         kernel_frontier_coalesce_ptr_pc<<<numblocks, numthreads>>>((unsigned int*)label_d, (unsigned int) level, vertex_count, vertexList_d, h_array->d_array_ptr, active,(unsigned long long int*)changed_d, curr_frontier_d, next_frontier_d);
                         tmp_front = curr_frontier_d;
                         curr_frontier_d = next_frontier_d;
                         next_frontier_d = tmp_front;
                         break;
                     
                     default:
                         fprintf(stderr, "Invalid type\n");
                         exit(1);
                         break;
                 }

                 iter++;
                 level++;

                 cuda_err_chk(hipMemcpy(&changed_h, changed_d, sizeof(uint64_t), hipMemcpyDeviceToHost));
                 //auto end = std::chrono::system_clock::now();
                 // if(mem == BAFS_DIRECT) {
                 //     h_array->print_reset_stats();

                 // }
                 //auto elapsed = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
                 //std::cout << "Iter "<< iter << " Time: " << elapsed.count() << " ms" << std::endl;

                 //break;
             } while(changed_h);

             cuda_err_chk(hipEventRecord(end, 0));
             cuda_err_chk(hipEventSynchronize(end));
             cuda_err_chk(hipEventElapsedTime(&milliseconds, start, end));
             if(iter > 1){
                 printf("run %*d: ", 3, i);
                 printf("src %*u, ", 10, src);
                 printf("iteration %*u, ", 3, iter);
                 printf("time %*f ms\n", 12, milliseconds);
                 if(mem == BAFS_DIRECT) {
                     h_array->print_reset_stats();
                 }
                 fflush(stdout);
                 avg_milliseconds += (double)milliseconds;
				 num_run++; 
			 }
			 else {
                 avg_milliseconds += 0;
			 }
            
             if(settings.src == 0)
                   src += vertex_count / total_run;
             printf("\nBFS-%d Graph:%s \t Impl: %d \t SSD: %d \t Stride: %llu \t CL: %d \t AvgTime %f ms\n", i, filename.c_str(), type, settings.n_ctrls, settings.stride, settings.pageSize, avg_milliseconds / num_run);
         }
         
         free(vertexList_h);
         if((type==BASELINE_PC)||(type == COALESCE_PC)||(type == COALESCE_PTR_PC)||(type==COALESCE_HASH_PTR_PC) ||(type == COALESCE_CHUNK_PC)||(type==FRONTIER_BASELINE_PC)||(type == FRONTIER_COALESCE_PC)||(type==FRONTIER_COALESCE_PTR_PC)){
            delete h_pc; 
            delete h_range; 
            delete h_array;
         }
         if (edgeList_h)
             free(edgeList_h);
         cuda_err_chk(hipFree(vertexList_d));
         cuda_err_chk(hipFree(label_d));
         cuda_err_chk(hipFree(changed_d));

         cuda_err_chk(hipFree(globalvisitedcount_d));
         cuda_err_chk(hipFree(vertexVisitCount_d));
         vertexVisitCount_h.clear();

         if (edgeList_d)
             cuda_err_chk(hipFree(edgeList_d));
         
         for (size_t i = 0 ; i < settings.n_ctrls; i++)
             delete ctrls[i];
    }
    catch (const error& e){
        fprintf(stderr, "Unexpected error: %s\n", e.what());
        return 1;
    }
    return 0;
}
