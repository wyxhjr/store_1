#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <nvm_ctrl.h>
#include <nvm_types.h>
#include <nvm_queue.h>
#include <nvm_util.h>
#include <nvm_admin.h>
#include <nvm_error.h>
#include <nvm_cmd.h>
#include <string>
#include <stdexcept>
#include <vector>
#include <cstdio>
#include <cstdint>
#include <cstring>
#include <fcntl.h>
#include <unistd.h>
#include <sys/mman.h>
#include <ctrl.h>
#include <buffer.h>
#include "settings.h"
#include <event.h>
#include <queue.h>
#include <nvm_parallel_queue.h>
#include <nvm_io.h>
#include <page_cache.h>
#include <util.h>
#include <iostream>
#include <fstream>
#include <algorithm>
#include <algorithm>
#include <iostream>
#include <list>
#include <numeric>
#include <random>
#include <vector>
#include <map>
#include <fcntl.h>
#include <unistd.h>
#include <sys/mman.h>
#include <sys/stat.h>
#ifdef __DIS_CLUSTER__
#include <sisci_api.h>
#endif

using error = std::runtime_error;
using std::string;



//uint32_t n_ctrls = 1;
const char* const ctrls_paths[] = {"/dev/libnvm0", "/dev/libnvm1", "/dev/libnvm2", "/dev/libnvm3", "/dev/libnvm4", "/dev/libnvm5", "/dev/libnvm6", "/dev/libnvm7"};

__global__
void flush_kernel(page_cache_d_t* cache) {
    uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t page = tid;
    // if (tid == 0) {
    //     hexdump(cache->base_addr, 4096);
    // }
    if (page < cache->n_pages) {
        uint32_t v = cache->cache_pages[page].page_take_lock.load(simt::memory_order_acquire);
        if (v != FREE) {
            uint32_t previous_global_address = cache->cache_pages[page].page_translation;
            uint32_t previous_range = previous_global_address & cache->n_ranges_mask;
            uint32_t previous_address = previous_global_address >> cache->n_ranges_bits;
            uint32_t expected_state = cache->ranges[previous_range][previous_address].state.load(simt::memory_order_acquire);
            if (expected_state == VALID_DIRTY) {
                uint64_t ctrl = get_backing_ctrl_(previous_address, cache->n_ctrls, cache->ranges_dists[previous_range]);
                //uint64_t get_backing_page(const uint64_t page_start, const size_t page_offset, const uint64_t n_ctrls, const data_dist_t dist) {
                uint64_t index = get_backing_page_(cache->ranges_page_starts[previous_range], previous_address,
                                                   cache->n_ctrls, cache->ranges_dists[previous_range]);
                // printf("Eviciting range_id: %llu\tpage_id: %llu\tctrl: %llx\tindex: %llu\n",
                //        (unsigned long long) previous_range, (unsigned long long)previous_address,
                //        (unsigned long long) ctrl, (unsigned long long) index);
                if (ctrl == ALL_CTRLS) {
                    for (ctrl = 0; ctrl < cache->n_ctrls; ctrl++) {
                        Controller* c = cache->d_ctrls[ctrl];
                        uint32_t queue = (tid/32) % (c->n_qps);
                        write_data(cache, (c->d_qps)+queue, (index*cache->n_blocks_per_page), cache->n_blocks_per_page, page);
                    }
                }
                else {

                    Controller* c = cache->d_ctrls[ctrl];
                    uint32_t queue = (tid/32) % (c->n_qps);

                    //index = ranges_page_starts[previous_range] + previous_address;


                    write_data(cache, (c->d_qps)+queue, (index*cache->n_blocks_per_page), cache->n_blocks_per_page, page);
                }
            }
        }

    }
}
__global__
void sequential_access_kernel(array_d_t<unsigned long long int>* dr, uint64_t n_reqs, unsigned long long* req_count, uint64_t reqs_per_thread) {

    uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n_reqs) {
        for (size_t i = 0; i < reqs_per_thread; i++)
            req_count += (*dr)[(tid)];

    }

}

__global__
void random_access_kernel(array_d_t<unsigned long long int>* dr, uint64_t n_reqs, unsigned long long* req_count, uint64_t* assignment, uint64_t reqs_per_thread, unsigned long long int* f_in) {

    uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n_reqs) {
        for (size_t i = 0; i < reqs_per_thread; i++) {
            uint64_t idx = assignment[tid];
            unsigned long long int v = f_in[idx];
            dr->AtomicAdd(idx, v);
        }


    }

}

int main(int argc, char** argv) {

    Settings settings;
    try
    {
        settings.parseArguments(argc, argv);
    }
    catch (const string& e)
    {
        fprintf(stderr, "%s\n", e.c_str());
        fprintf(stderr, "%s\n", Settings::usageString(argv[0]).c_str());
        return 1;
    }


    hipDeviceProp_t properties;
    if (hipGetDeviceProperties(&properties, settings.cudaDevice) != hipSuccess)
    {
        fprintf(stderr, "Failed to get CUDA device properties\n");
        return 1;
    }

    try {
        uint64_t b_size = settings.blkSize;//64;
        uint64_t g_size = (settings.numThreads + b_size - 1)/b_size;//80*16;
        uint64_t n_threads = b_size * g_size;


        uint64_t page_size = settings.pageSize;
        uint64_t n_pages = settings.numPages;
        uint64_t total_cache_size = (page_size * n_pages);
        #define TYPE unsigned long long int
        uint64_t n_elems = settings.numThreads;
        uint64_t t_size = n_elems * sizeof(TYPE);
        const char* input_f;

        if(settings.input == nullptr){
            fprintf(stderr, "Input file required\n");
            return 1;
        }
        else {
            input_f = settings.input;
            printf("File is : %s\n",input_f);
        }
        //Controller ctrl(settings.controllerPath, settings.nvmNamespace, settings.cudaDevice);
        void* map_in;
        int fd_in;
        struct stat sb_in;

        if((fd_in = open(input_f, O_RDWR)) == -1){
            fprintf(stderr, "Input file cannot be opened\n");
            return 1;
        }

        fstat(fd_in, &sb_in);

        map_in = mmap(NULL, t_size, PROT_READ | PROT_WRITE, MAP_SHARED | MAP_LOCKED, fd_in, 0);

        if((map_in == (void*)-1)){
                fprintf(stderr,"Input file map failed %d\n",map_in);
                return 1;
        }

        cuda_err_chk(hipSetDevice(settings.cudaDevice));

        unsigned long long int* f_in_d;
        cuda_err_chk(hipHostRegister(map_in, t_size, hipHostRegisterDefault));
        cuda_err_chk(hipHostGetDevicePointer(&f_in_d, map_in, 0));

        std::vector<Controller*> ctrls(settings.n_ctrls);
        for (size_t i = 0 ; i < settings.n_ctrls; i++)
            ctrls[i] = new Controller(ctrls_paths[i], settings.nvmNamespace, settings.cudaDevice, settings.queueDepth, settings.numQueues);
        return 1;

        //auto dma = createDma(ctrl.ctrl, NVM_PAGE_ALIGN(64*1024*10, 1UL << 16), settings.cudaDevice, settings.adapter, settings.segmentId);

        //std::cout << dma.get()->vaddr << std::endl;
        //QueuePair h_qp(ctrl, settings, 1);
        //std::cout << "in main: " << std::hex << h_qp.sq.cid << "raw: " << h_qp.sq.cid<< std::endl;
        //std::memset(&h_qp, 0, sizeof(QueuePair));
        //prepareQueuePair(h_qp, ctrl, settings, 1);
        //const uint32_t ps, const uint64_t np, const uint64_t c_ps, const Settings& settings, const Controller& ctrl)
        //
        /*
        Controller** d_ctrls;
        cuda_err_chk(hipMalloc(&d_ctrls, n_ctrls*sizeof(Controller*)));
        for (size_t i = 0; i < n_ctrls; i++)
            cuda_err_chk(hipMemcpy(d_ctrls+i, &(ctrls[i]->d_ctrl), sizeof(Controller*), hipMemcpyHostToDevice));
        */

        //uint64_t n_pages = total_cache_size/page_size;


        page_cache_t h_pc(page_size, n_pages, settings.cudaDevice, ctrls[0][0], (uint64_t) 64, ctrls);
        std::cout << "finished creating cache\n";

        //QueuePair* d_qp;
        page_cache_d_t* d_pc = (h_pc.d_pc_ptr);


        range_t<unsigned long long int> h_range((uint64_t)0, (uint64_t)n_elems, (uint64_t)0, (uint64_t)((t_size+page_size-1)/page_size), (uint64_t)0, (uint64_t)page_size, &h_pc, settings.cudaDevice);
        range_t<unsigned long long int>* d_range = (range_t<unsigned long long int>*) h_range.d_range_ptr;

        std::vector<range_t<unsigned long long int>*> vr(1);
        vr[0] = & h_range;
        //(const uint64_t num_elems, const uint64_t disk_start_offset, const std::vector<range_t<T>*>& ranges, Settings& settings)
        array_t<unsigned long long int> a(n_elems, 0, vr, settings.cudaDevice);


        std::cout << "finished creating range\n";




        unsigned long long* d_req_count;
        cuda_err_chk(hipMalloc(&d_req_count, sizeof(unsigned long long)));
        cuda_err_chk(hipMemset(d_req_count, 0, sizeof(unsigned long long)));
        std::cout << "atlaunch kernel\n";
        char st[15];
        cuda_err_chk(hipDeviceGetPCIBusId(st, 15, settings.cudaDevice));
        std::cout << st << std::endl;
        std::cout << std::dec << "File Size: " << t_size << " Num Elems In File: " <<
            (t_size/sizeof(TYPE)) << " Num elems in vec: " << n_threads << std::endl;
        //uint64_t* assignment;
        uint64_t* d_assignment;
        if (settings.random) {
            std::vector<uint64_t> assignment(n_threads);
            uint64_t start = 0;
            std::iota(assignment.begin(), assignment.end(), start);
            std::cout << "Finished iota\n";
            // for (size_t i = 0; i < assignment.size(); i++) {
            //     assignment[i] = start++;
            // }
            std::cout <<std::dec << "last vec elem i : " << assignment.size()-1 << " val: " << assignment[assignment.size()-1] << std::endl;
            // for (size_t i = 0; i < assignment.size(); i++) {
            //     if (assignment[i] >= n_threads)
            //         std::cout <<std::dec << "i : " << i << " val: " << assignment[i] << std::endl;
            // }
            std::cout << std::dec << "max elem: " << *std::max_element(assignment.begin(), assignment.end()) << std::endl;
            std::shuffle(assignment.begin(), assignment.end(), std::mt19937_64{std::random_device{}()});
            std::cout << "Finished shuffle\n";

            cuda_err_chk(hipMallocManaged(&d_assignment, n_threads*sizeof(uint64_t)));
            cuda_err_chk(hipMemcpy(d_assignment, assignment.data(),  n_threads*sizeof(uint64_t), hipMemcpyHostToDevice));
        }
        //return 1;
        Event before;
        //access_kernel<<<g_size, b_size>>>(h_pc.d_ctrls, d_pc, page_size, n_threads, d_req_count, settings.n_ctrls, d_assignment, settings.numReqs);
        if (settings.random)
            random_access_kernel<<<g_size, b_size>>>(a.d_array_ptr, n_threads, d_req_count, d_assignment, settings.numReqs, f_in_d);
        cuda_err_chk(hipDeviceSynchronize());
        std::cout << "Finished Random access kernel\n";

        flush_kernel<<<n_pages, 1>>>(d_pc);

        //new_kernel<<<1,1>>>();
        //uint8_t* ret_array = (uint8_t*) malloc(n_pages*page_size);

        //cuda_err_chk(hipMemcpy(ret_array, h_pc.base_addr,page_size*n_pages, hipMemcpyDeviceToHost));
        cuda_err_chk(hipDeviceSynchronize());


        double elapsed = 0;
        uint64_t ios = g_size*b_size*settings.numReqs;
        uint64_t data = ios*sizeof(uint64_t);
        double iops = ((double)ios)/(elapsed/1000000);
        double bandwidth = (((double)data)/(elapsed/1000000))/(1024ULL*1024ULL*1024ULL);
        a.print_reset_stats();
        std::cout << std::dec << "Elapsed Time: " << elapsed << "\tNumber of Read Ops: "<< ios << "\tData Size (bytes): " << data << std::endl;
        std::cout << std::dec << "Read Ops/sec: " << iops << "\tEffective Bandwidth(GB/S): " << bandwidth << std::endl;

        //std::cout << std::dec << ctrls[0]->ns.lba_data_size << std::endl;

        //std::ofstream ofile("../data", std::ios::binary | std::ios::trunc);
        //ofile.write((char*)ret_array, data);
        //ofile.close();

        for (size_t i = 0 ; i < settings.n_ctrls; i++)
            delete ctrls[i];
        //hexdump(ret_array, n_pages*page_size);
/*
        hipFree(d_qp);
        hipFree(d_pc);
        hipFree(d_req_count);
        free(ret_array);
*/

        //std::cout << "END\n";

        //std::cout << RAND_MAX << std::endl;

    }
    catch (const error& e) {
        fprintf(stderr, "Unexpected error: %s\n", e.what());
        return 1;
    }



}
