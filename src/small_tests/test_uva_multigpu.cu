
#include <hip/hip_runtime.h>
#include <iostream>

#define CUDA_CHECK(val) \
  { nv::cuda_check_((val), __FILE__, __LINE__); }

namespace nv {

class CudaException : public std::runtime_error {
 public:
  CudaException(const std::string& what) : runtime_error(what) {}
};

inline void cuda_check_(hipError_t val, const char* file, int line) {
  if (val != hipSuccess) {
    throw CudaException(std::string(file) + ":" + std::to_string(line) +
                        ": CUDA error " + std::to_string(val) + ": " +
                        hipGetErrorString(val));
  }
}
}  // namespace nv

__global__ void get_and_set_kernel(int* d_a) {
  printf("In card 1: d_a = %d\n", *d_a);
  *d_a = 4321;
}

int main() {
  int* dev_a;
  int host_a = 1234;
  int size = 4;

  int can_access_peer_0_1 = true;
  hipDeviceCanAccessPeer(&can_access_peer_0_1, 0, 1);
  if (!can_access_peer_0_1) {
    std::cerr << "can not access p2p";
    // std::exit(-1);
  }

  hipSetDevice(1);
  // 这行是必须的
  hipDeviceEnablePeerAccess(0, 0);

  hipSetDevice(0);
  hipMalloc((void**)&dev_a, size);
  hipMemcpy(dev_a, &host_a, size, hipMemcpyHostToDevice);
  CUDA_CHECK(hipGetLastError());
  //
  hipSetDevice(1);
  get_and_set_kernel<<<1, 1>>>(dev_a);
  hipDeviceSynchronize();
  CUDA_CHECK(hipGetLastError());
  //
  hipSetDevice(0);
  hipMemcpy(&host_a, dev_a, size, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  std::cout << "In card 0: d_a = " << host_a << "\n";
  CUDA_CHECK(hipGetLastError());

  return 0;
}