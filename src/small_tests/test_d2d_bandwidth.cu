#include <hip/hip_runtime.h>
#include <stdio.h>

#include <chrono>
#include <iostream>

#define SIZE (4 * 1024 * 1024 * 1024LL)

int main() {
    hipSetDevice(0);
  float* deviceData1;

  hipMalloc((void**)&deviceData1, SIZE * sizeof(float)
                 );

    hipSetDevice(1);
  float* deviceData2;
  hipMalloc((void**)&deviceData2, SIZE * sizeof(float));

    hipSetDevice(0);
  std::chrono::steady_clock::time_point t1 = std::chrono::steady_clock::now();
  hipMemcpy(deviceData2, deviceData1, SIZE * sizeof(float),
             hipMemcpyDeviceToDevice);
  hipDeviceSynchronize();
  std::chrono::steady_clock::time_point t2 = std::chrono::steady_clock::now();

  std::cout
      << (double)SIZE / (1024 * 1024 * 1024) * sizeof(float) /
             (double)std::chrono::duration_cast<std::chrono::seconds>(t2 - t1).count()
      << " GB/s";

  return 0;
}
