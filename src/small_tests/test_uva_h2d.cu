#include <hip/hip_runtime.h>

#include <hiprand/hiprand_kernel.h>

#include <iostream>

const int cycle = 1;

__global__ void generatePattern(int *pos, int size) {
  hiprandState state;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int warp_id = tid / 32;
  int lane = tid % 32;
  hiprand_init(warp_id, 0, 0, &state);
  int *pos_off = pos + warp_id * cycle;
  if (lane == 0) {
    for (int i = 0; i < cycle; i++) {
      int index = hiprand(&state) % size;
      pos_off[i] = index;
    }
  }
}

// __global__ void randomAccessKernel(double4 *data, int size, double4 *output,
//                                    int *pos) {
//   int tid = blockIdx.x * blockDim.x + threadIdx.x;
//   int warp_id = tid / 32;
//   int lane = tid % 32;
//   int *pos_off = pos + warp_id * cycle;
//   for (int i = 0; i < cycle; i++) {
//     int index = pos_off[i];
//     output[index + lane] = data[index + lane];
//   }
// }

__global__ void randomAccessKernel(float *src, int size_dim, float *dst) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int warp_id = tid / 32;
  int lane = tid % 32;

  if (tid < size_dim) dst[tid] = src[tid];
}

int main() {
  const int start_dim = 1 * 1024 / sizeof(float);
  const int end_dim = 128 * 1024 * 1024 / sizeof(float);

  float *hostData, *deviceOutput;
  hipHostMalloc(&hostData, end_dim * sizeof(float), hipHostMallocDefault);
  hipMalloc(&deviceOutput, end_dim * sizeof(float));

  for (int nr_dim = start_dim; nr_dim <= end_dim; nr_dim *= 2) {
    const int block_size = 256;
    const int block_cnt = nr_dim / block_size + 1;

    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
      std::cout << "Error: " << hipGetErrorString(err) << std::endl;
      return -1;
    }
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    randomAccessKernel<<<block_cnt, block_size>>>(hostData, nr_dim,
                                                  deviceOutput);
    err = hipEventRecord(stop);
    hipError_t err2 = hipEventSynchronize(stop);

    if (err != hipSuccess || err2 != hipSuccess) {
      std::cout << "Error: " << hipGetErrorString(err) << std::endl;
      std::cout << "Error: " << hipGetErrorString(err2) << std::endl;
    }
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << nr_dim * sizeof(float) << "\t"
              << sizeof(float) * nr_dim / milliseconds / 1000000 << " GB/s"
              << std::endl;
  }

  return 0;
}
