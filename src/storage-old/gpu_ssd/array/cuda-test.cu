
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>

int main(int argc, char** argv){
 int deviceCount;
 hipError_t err = hipGetDeviceCount(&deviceCount);
 std::cout << "device Count = " << deviceCount;
 if (err != hipSuccess)
 {
     std::cout <<  "main, cudaGetDeviceCount:" << hipGetErrorString(err) <<std::endl << std::flush;
     throw std::string("Unexpected error: ") + hipGetErrorString(err);
 }
 return 0;
}